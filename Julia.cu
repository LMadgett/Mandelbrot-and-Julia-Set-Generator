
#include <hip/hip_runtime.h>
extern "C"
{
	//Device code
	__global__ void Julia(const int pixWidth, const int pixHeight, const float minX, const float maxX,
	const float minY, const float maxY, float xInc, float yInc, const float cReal, const float cImg, const int maxIter, int* counts, int N)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if(i < N)
		{
			int xPos = i % pixWidth;
			int yPos = i / pixWidth;
			float zReal = minX + (xPos * xInc);
			float zImg = maxY - (yPos * yInc);
			int iter = 0;
			while((zReal * zReal) + (zImg * zImg) <= 4.0 && (iter < maxIter))
			{
				float nextZReal = (zReal * zReal) - (zImg * zImg) + cReal;
				float nextZImg = (2.0 * zReal * zImg) + cImg;
				zReal = nextZReal;
				zImg = nextZImg;
				iter = iter + 1;
			}
			counts[i] = iter;
		}
	}
}